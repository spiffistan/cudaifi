#include "hip/hip_runtime.h"
#include <stdio.h>
#include <inttypes.h>
#include <stdint.h>
#include "c63.h"
#include "cuda_util.hcu"
#include "workqueue.h"
void catchCudaError(const char *message) {
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf(stderr, "ERROR: %s: %s\n", message, hipGetErrorString(error));
		exit(-1);
	}
}
struct cuda_frame *cframe;

extern "C" void cuda_init(c63_common *cm) {

	cframe = (cuda_frame*) malloc(sizeof(cuda_frame));
	cframe->image = (yuv_t*) malloc(sizeof(yuv_t));
	cframe->curr_recons = (yuv_t*) malloc(sizeof(yuv_t));
	cframe->last_recons = (yuv_t*) malloc(sizeof(yuv_t));
	cframe->predicted = (yuv_t*) malloc(sizeof(yuv_t));
	cframe->residuals = (dct_t*) malloc(sizeof(dct_t));

	cframe->mb_width_Y = cm->mb_cols;
	cframe->mb_height_Y = cm->mb_rows;
	cframe->mb_width_UV = cm->mb_cols / 2;
	cframe->mb_height_UV = cm->mb_rows / 2;

	cframe->dct_threadDim = dim3(8, 8, 4);
	cframe->dct_blockDim_Y = dim3(ceil(cm->mb_cols / cframe->dct_threadDim.z), cm->mb_rows);
	cframe->dct_blockDim_UV = dim3(ceil((cm->vpw / 8.0f) / cframe->dct_threadDim.z), cm->vph / 8);

	cframe->mc_threadDim = dim3(8, 8, 4);
	cframe->mc_blockDim_Y = dim3(ceil(cm->mb_cols / cframe->mc_threadDim.z), cm->mb_rows);
	cframe->mc_blockDim_UV = dim3(ceil((cm->vpw / 8.0f) / cframe->mc_threadDim.z), cm->vph / 8);

	cframe->me_threadDim = dim3(32, 16, 1);
	cframe->me_blockDim_Y = dim3(cframe->mb_width_Y, cframe->mb_height_Y);
	cframe->me_blockDim_UV = dim3(cframe->mb_width_UV, cframe->mb_height_UV);

	hipMallocPitch(&cframe->image->Y, &cframe->image_pitch[0], cm->ypw, cm->yph);
	hipMallocPitch(&cframe->image->U, &cframe->image_pitch[1], cm->upw, cm->uph);
	hipMallocPitch(&cframe->image->V, &cframe->image_pitch[2], cm->vpw, cm->vph);

	hipMallocPitch(&cframe->curr_recons->Y, &cframe->curr_recons_pitch[0], cm->ypw, cm->yph);
	hipMallocPitch(&cframe->curr_recons->U, &cframe->curr_recons_pitch[1], cm->ypw, cm->yph);
	hipMallocPitch(&cframe->curr_recons->V, &cframe->curr_recons_pitch[2], cm->ypw, cm->yph);

	hipMallocPitch(&cframe->last_recons->Y, &cframe->last_recons_pitch[0], cm->ypw, cm->yph);
	hipMallocPitch(&cframe->last_recons->U, &cframe->last_recons_pitch[1], cm->ypw, cm->yph);
	hipMallocPitch(&cframe->last_recons->V, &cframe->last_recons_pitch[2], cm->ypw, cm->yph);

	hipMalloc(&cframe->predicted->Y, cm->ypw * cm->yph);
	hipMalloc(&cframe->predicted->U, cm->upw * cm->uph);
	hipMalloc(&cframe->predicted->V, cm->vpw * cm->vph);

	hipMalloc(&cframe->residuals->Ydct, cm->ypw * cm->yph * sizeof(dct_t));
	hipMalloc(&cframe->residuals->Udct, cm->vpw * cm->vph * sizeof(dct_t));
	hipMalloc(&cframe->residuals->Vdct, cm->upw * cm->uph * sizeof(dct_t));

	hipMalloc(&cframe->mbs[0], cframe->mb_width_Y * cframe->mb_height_Y * sizeof(macroblock));
	hipMalloc(&cframe->mbs[1], cframe->mb_width_UV * cframe->mb_height_UV * sizeof(macroblock));
	hipMalloc(&cframe->mbs[2], cframe->mb_width_UV * cframe->mb_height_UV * sizeof(macroblock));

	hipMalloc(&cframe->qtables[0], 64 * sizeof(uint8_t));
	hipMalloc(&cframe->qtables[1], 64 * sizeof(uint8_t));
	hipMalloc(&cframe->qtables[2], 64 * sizeof(uint8_t));

	hipMemcpy(cframe->qtables[0], cm->quanttbl[0], 64 * sizeof(uint8_t), hipMemcpyHostToDevice);
	hipMemcpy(cframe->qtables[1], cm->quanttbl[1], 64 * sizeof(uint8_t), hipMemcpyHostToDevice);
	hipMemcpy(cframe->qtables[2], cm->quanttbl[2], 64 * sizeof(uint8_t), hipMemcpyHostToDevice);
	catchCudaError("CUDA_INIT");

}

void cuda_new_frame(c63_common *cm, workitem_t *work) {

	yuv_t *tmp = cframe->last_recons;
	cframe->last_recons = cframe->curr_recons;
	cframe->curr_recons = tmp;

	size_t pitch[3];
	pitch[0] = cframe->last_recons_pitch[0];
	pitch[1] = cframe->last_recons_pitch[1];
	pitch[2] = cframe->last_recons_pitch[2];
	cframe->last_recons_pitch[0] = cframe->curr_recons_pitch[0];
	cframe->last_recons_pitch[1] = cframe->curr_recons_pitch[1];
	cframe->last_recons_pitch[2] = cframe->curr_recons_pitch[2];
	cframe->curr_recons_pitch[0] = pitch[0];
	cframe->curr_recons_pitch[1] = pitch[1];
	cframe->curr_recons_pitch[2] = pitch[2];

	hipMemset(cframe->residuals->Ydct, 0, cm->yph * cm->ypw * sizeof(int16_t));
	hipMemset(cframe->residuals->Udct, 0, cm->yph * cm->ypw * sizeof(int16_t));
	hipMemset(cframe->residuals->Vdct, 0, cm->yph * cm->ypw * sizeof(int16_t));

	hipMemset2D(cframe->curr_recons->Y, cframe->curr_recons_pitch[0], 0, cm->ypw, cm->yph);
	hipMemset2D(cframe->curr_recons->U, cframe->curr_recons_pitch[1], 0, cm->ypw, cm->yph);
	hipMemset2D(cframe->curr_recons->V, cframe->curr_recons_pitch[2], 0, cm->ypw, cm->yph);

	hipMemset(cframe->predicted->Y, 0, cm->ypw * cm->yph);
	hipMemset(cframe->predicted->U, 0, cm->upw * cm->uph);
	hipMemset(cframe->predicted->V, 0, cm->vpw * cm->vph);

	hipMemset(cframe->mbs[0], 0, cframe->mb_width_Y * cframe->mb_height_Y * sizeof(macroblock));
	hipMemset(cframe->mbs[1], 0, cframe->mb_width_UV * cframe->mb_height_UV * sizeof(macroblock));
	hipMemset(cframe->mbs[2], 0, cframe->mb_width_UV * cframe->mb_height_UV * sizeof(macroblock));

	hipMemcpy2D(cframe->image->Y, cframe->image_pitch[0], work->image->Y, cm->width, cm->width, cm->height, hipMemcpyHostToDevice);
	hipMemcpy2D(cframe->image->U, cframe->image_pitch[1], work->image->U, cm->width/2, cm->width/2, cm->height, hipMemcpyHostToDevice);
	hipMemcpy2D(cframe->image->V, cframe->image_pitch[2], work->image->V, cm->width/2, cm->width/2, cm->height, hipMemcpyHostToDevice);

	catchCudaError("CUDA_NEW_FRAME");
}

void cuda_store_values(struct c63_common *cm, workitem_t *work) {
	hipMemcpy(work->mbs[0], cframe->mbs[0], cframe->mb_width_Y * cframe->mb_height_Y * sizeof(macroblock), hipMemcpyDeviceToHost);
	hipMemcpy(work->mbs[1], cframe->mbs[1], cframe->mb_width_UV * cframe->mb_height_UV * sizeof(macroblock), hipMemcpyDeviceToHost);
	hipMemcpy(work->mbs[2], cframe->mbs[2], cframe->mb_width_UV * cframe->mb_height_UV * sizeof(macroblock), hipMemcpyDeviceToHost);

	hipMemcpy(work->residuals->Ydct, cframe->residuals->Ydct, cm->ypw * cm->yph * sizeof(int16_t), hipMemcpyDeviceToHost);
	hipMemcpy(work->residuals->Udct, cframe->residuals->Udct, cm->upw * cm->uph * sizeof(int16_t), hipMemcpyDeviceToHost);
	hipMemcpy(work->residuals->Vdct, cframe->residuals->Vdct, cm->vpw * cm->vph * sizeof(int16_t), hipMemcpyDeviceToHost);
	catchCudaError("CUDA_STORE_VALUES");

}
extern "C" void cuda_run(struct c63_common *cm, workitem_t *work) {
	cuda_new_frame(cm, work);

	if (!work->keyframe) {
		/* Motion Estimation and compensation */
		c63_motion_estimate(cm, cframe);

	}

	/* DCT and Quantization */
	dct_quantize_frame(cm, cframe);
	idct_dequantize_frame(cm, cframe);

	cuda_store_values(cm, work);
}

extern "C" void cuda_stop() {
	hipFree(cframe->image->Y);
	hipFree(cframe->image->U);
	hipFree(cframe->image->V);
	hipFree(cframe->curr_recons->Y);
	hipFree(cframe->curr_recons->U);
	hipFree(cframe->curr_recons->V);
	hipFree(cframe->last_recons->Y);
	hipFree(cframe->last_recons->U);
	hipFree(cframe->last_recons->V);
	hipFree(cframe->predicted->Y);
	hipFree(cframe->predicted->U);
	hipFree(cframe->predicted->V);
	hipFree(cframe->residuals->Ydct);
	hipFree(cframe->residuals->Udct);
	hipFree(cframe->residuals->Vdct);

	free(cframe->image);
	free(cframe->predicted);
	free(cframe->curr_recons);
	free(cframe->last_recons);
	free(cframe->residuals);
	free(cframe);
}
