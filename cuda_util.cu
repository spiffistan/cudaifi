#include "cuda_util.hcu"
#include <stdio.h>
void catchCudaError(const char *message)
{
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "ERROR: %s: %s\n", message, hipGetErrorString(error));
        exit(-1);
    }
}

