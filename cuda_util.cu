#include "hip/hip_runtime.h"
#include <stdio.h>
#include <inttypes.h>
#include <stdint.h>
#include "c63.h"
#include "cuda_util.hcu"

void catchCudaError(const char *message) {
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf(stderr, "ERROR: %s: %s\n", message, hipGetErrorString(error));
		exit(-1);
	}
}
struct cuda_frame *cframe;

extern "C" void cuda_init(c63_common *cm) {

	cframe = (cuda_frame*) malloc(sizeof(cuda_frame));
	cframe->image = (yuv_t*) malloc(sizeof(yuv_t));
	cframe->curr_recons = (yuv_t*) malloc(sizeof(yuv_t));
	cframe->last_recons = (yuv_t*) malloc(sizeof(yuv_t));
	cframe->predicted = (yuv_t*) malloc(sizeof(yuv_t));
	cframe->residuals = (dct_t*) malloc(sizeof(dct_t));

	cframe->mb_width_Y = cm->mb_cols;
	cframe->mb_height_Y = cm->mb_rows;
	cframe->mb_width_UV = cm->mb_cols / 2;
	cframe->mb_height_UV = cm->mb_rows / 2;

	cframe->dct_threadDim = dim3(8, 8, 4);
	cframe->dct_blockDim_Y = dim3(ceil(cm->mb_cols / cframe->dct_threadDim.z), cm->mb_rows);
	cframe->dct_blockDim_UV = dim3(ceil((cm->vpw / 8.0f) / cframe->dct_threadDim.z), cm->vph / 8);

	cframe->mc_threadDim = dim3(8, 8, 4);
	cframe->mc_blockDim_Y = dim3(ceil(cm->mb_cols / cframe->mc_threadDim.z), cm->mb_rows);
	cframe->mc_blockDim_UV = dim3(ceil((cm->vpw / 8.0f) / cframe->mc_threadDim.z), cm->vph / 8);

	cframe->me_threadDim = dim3(32, 16, 1);
	cframe->me_blockDim_Y = dim3(cframe->mb_width_Y, cframe->mb_height_Y);
	cframe->me_blockDim_UV = dim3(cframe->mb_width_UV, cframe->mb_height_UV);

	hipMallocPitch(&cframe->image->Y, &cframe->image_pitch[0], cm->ypw, cm->yph);
	hipMallocPitch(&cframe->image->U, &cframe->image_pitch[1], cm->upw, cm->uph);
	hipMallocPitch(&cframe->image->V, &cframe->image_pitch[2], cm->vpw, cm->vph);

	hipMallocPitch(&cframe->curr_recons->Y, &cframe->curr_recons_pitch[0], cm->ypw, cm->yph);
	hipMallocPitch(&cframe->curr_recons->U, &cframe->curr_recons_pitch[1], cm->ypw, cm->yph);
	hipMallocPitch(&cframe->curr_recons->V, &cframe->curr_recons_pitch[2], cm->ypw, cm->yph);

	hipMallocPitch(&cframe->last_recons->Y, &cframe->last_recons_pitch[0], cm->ypw, cm->yph);
	hipMallocPitch(&cframe->last_recons->U, &cframe->last_recons_pitch[1], cm->ypw, cm->yph);
	hipMallocPitch(&cframe->last_recons->V, &cframe->last_recons_pitch[2], cm->ypw, cm->yph);

	hipMallocPitch(&cframe->predicted->Y, &cframe->predicted_pitch[0], cm->ypw, cm->yph);
	hipMallocPitch(&cframe->predicted->U, &cframe->predicted_pitch[1], cm->upw, cm->uph);
	hipMallocPitch(&cframe->predicted->V, &cframe->predicted_pitch[2], cm->vpw, cm->vph);

	hipMalloc(&cframe->residuals->Ydct, cm->ypw * cm->yph * sizeof(dct_t));
	hipMalloc(&cframe->residuals->Udct, cm->vpw * cm->vph * sizeof(dct_t));
	hipMalloc(&cframe->residuals->Vdct, cm->upw * cm->uph * sizeof(dct_t));

	hipMalloc(&cframe->mbs[0], cframe->mb_width_Y * cframe->mb_height_Y * sizeof(macroblock));
	hipMalloc(&cframe->mbs[1], cframe->mb_width_UV * cframe->mb_height_UV * sizeof(macroblock));
	hipMalloc(&cframe->mbs[2], cframe->mb_width_UV * cframe->mb_height_UV * sizeof(macroblock));

	hipMalloc(&cframe->qtables[0], 64 * sizeof(uint8_t));
	hipMalloc(&cframe->qtables[1], 64 * sizeof(uint8_t));
	hipMalloc(&cframe->qtables[2], 64 * sizeof(uint8_t));

	hipMemcpy(cframe->qtables[0], cm->quanttbl[0], 64 * sizeof(uint8_t), hipMemcpyHostToDevice);
	hipMemcpy(cframe->qtables[1], cm->quanttbl[1], 64 * sizeof(uint8_t), hipMemcpyHostToDevice);
	hipMemcpy(cframe->qtables[2], cm->quanttbl[2], 64 * sizeof(uint8_t), hipMemcpyHostToDevice);
	catchCudaError("CUDA_INIT");

}

void cuda_new_frame(c63_common *cm) {

	yuv_t *tmp = cframe->last_recons;
	cframe->last_recons = cframe->curr_recons;
	cframe->curr_recons = tmp;

	size_t pitch[3];
	pitch[0] = cframe->last_recons_pitch[0];
	pitch[1] = cframe->last_recons_pitch[1];
	pitch[2] = cframe->last_recons_pitch[2];
	cframe->last_recons_pitch[0] = cframe->curr_recons_pitch[0];
	cframe->last_recons_pitch[1] = cframe->curr_recons_pitch[1];
	cframe->last_recons_pitch[2] = cframe->curr_recons_pitch[2];
	cframe->curr_recons_pitch[0] = pitch[0];
	cframe->curr_recons_pitch[1] = pitch[1];
	cframe->curr_recons_pitch[2] = pitch[2];

	hipMemset(cframe->residuals->Ydct, 0, cm->yph * cm->ypw * sizeof(int16_t));
	hipMemset(cframe->residuals->Udct, 0, cm->yph * cm->ypw * sizeof(int16_t));
	hipMemset(cframe->residuals->Vdct, 0, cm->yph * cm->ypw * sizeof(int16_t));

	hipMemset2D(cframe->curr_recons->Y, cframe->curr_recons_pitch[0], 0, cm->ypw, cm->yph);
	hipMemset2D(cframe->curr_recons->U, cframe->curr_recons_pitch[1], 0, cm->ypw, cm->yph);
	hipMemset2D(cframe->curr_recons->V, cframe->curr_recons_pitch[2], 0, cm->ypw, cm->yph);

	hipMemset2D(cframe->predicted->Y, cframe->predicted_pitch[0], 0, cm->ypw, cm->yph);
	hipMemset2D(cframe->predicted->U, cframe->predicted_pitch[1], 0, cm->upw, cm->uph);
	hipMemset2D(cframe->predicted->V, cframe->predicted_pitch[2], 0, cm->vpw, cm->vph);

	hipMemset(cframe->mbs[0], 0, cframe->mb_width_Y * cframe->mb_height_Y * sizeof(macroblock));
	hipMemset(cframe->mbs[1], 0, cframe->mb_width_UV * cframe->mb_height_UV * sizeof(macroblock));
	hipMemset(cframe->mbs[2], 0, cframe->mb_width_UV * cframe->mb_height_UV * sizeof(macroblock));

	hipMemcpy2D(cframe->image->Y, cframe->image_pitch[0], cm->curframe->orig->Y, cm->ypw, cm->ypw, cm->yph, hipMemcpyHostToDevice);
	hipMemcpy2D(cframe->image->U, cframe->image_pitch[1], cm->curframe->orig->U, cm->upw, cm->upw, cm->uph, hipMemcpyHostToDevice);
	hipMemcpy2D(cframe->image->V, cframe->image_pitch[2], cm->curframe->orig->V, cm->vpw, cm->vpw, cm->vph, hipMemcpyHostToDevice);

	catchCudaError("CUDA_NEW_FRAME");
}

void cuda_store_values(struct c63_common *cm) {
	hipMemcpy(cm->curframe->mbs[0], cframe->mbs[0], cframe->mb_width_Y * cframe->mb_height_Y * sizeof(macroblock), hipMemcpyDeviceToHost);
	hipMemcpy(cm->curframe->mbs[1], cframe->mbs[1], cframe->mb_width_UV * cframe->mb_height_UV * sizeof(macroblock), hipMemcpyDeviceToHost);
	hipMemcpy(cm->curframe->mbs[2], cframe->mbs[2], cframe->mb_width_UV * cframe->mb_height_UV * sizeof(macroblock), hipMemcpyDeviceToHost);

	hipMemcpy(cm->curframe->residuals->Ydct, cframe->residuals->Ydct, cm->ypw * cm->yph * sizeof(int16_t), hipMemcpyDeviceToHost);
	hipMemcpy(cm->curframe->residuals->Udct, cframe->residuals->Udct, cm->upw * cm->uph * sizeof(int16_t), hipMemcpyDeviceToHost);
	hipMemcpy(cm->curframe->residuals->Vdct, cframe->residuals->Vdct, cm->vpw * cm->vph * sizeof(int16_t), hipMemcpyDeviceToHost);
	catchCudaError("CUDA_STORE_VALUES");

}
extern "C" void cuda_run(struct c63_common *cm) {
	cuda_new_frame(cm);

	if (!cm->curframe->keyframe) {
		/* Motion Estimation */
		c63_motion_estimate(cm, cframe);

		/* Motion Compensation */
		//motion_compensate_cuda(cm, cframe);

	}

	//cm->curframe->keyframe = 1;
	/* DCT and Quantization */
	dct_quantize_frame(cm, cframe);
	idct_dequantize_frame(cm, cframe);

	cuda_store_values(cm);
}

extern "C" void cuda_stop() {
	hipFree(cframe->image->Y);
	hipFree(cframe->image->U);
	hipFree(cframe->image->V);
	hipFree(cframe->curr_recons->Y);
	hipFree(cframe->curr_recons->U);
	hipFree(cframe->curr_recons->V);
	hipFree(cframe->last_recons->Y);
	hipFree(cframe->last_recons->U);
	hipFree(cframe->last_recons->V);
	hipFree(cframe->predicted->Y);
	hipFree(cframe->predicted->U);
	hipFree(cframe->predicted->V);
	hipFree(cframe->residuals->Ydct);
	hipFree(cframe->residuals->Udct);
	hipFree(cframe->residuals->Vdct);

	free(cframe->image);
	free(cframe->predicted);
	free(cframe->curr_recons);
	free(cframe->last_recons);
	free(cframe->residuals);
	free(cframe);
}
