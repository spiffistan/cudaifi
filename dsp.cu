#include "hip/hip_runtime.h"

#include <inttypes.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include "tables.h"
#include "dsp.hcu"

#define ISQRT2 0.70710678118654f
extern "C" {
static void transpose_block(float *in_data, float *out_data)
{
    int i,j;
    for (i=0; i<8; ++i)
        for (j=0; j<8; ++j)
        {
            out_data[i*8+j] = in_data[j*8+i];
        }
}

static void dct_1d(float *in_data, float *out_data)
{
    int i,j;

    for (j=0; j<8; ++j)
    {
        float dct = 0;

        for (i=0; i<8; ++i)
        {
            dct += in_data[i] * dctlookup[i][j];
        }

        out_data[j] = dct;
    }
}

static void idct_1d(float *in_data, float *out_data)
{
    int i,j;

    for (j=0; j<8; ++j)
    {
        float idct = 0;

        for (i=0; i<8; ++i)
        {
            idct += in_data[i] * dctlookup[j][i];
        }

        out_data[j] = idct;
    }
}


static void scale_block(float *in_data, float *out_data)
{
    int u,v;

    for (v=0; v<8; ++v)
    {
        for (u=0; u<8; ++u)
        {
            float a1 = !u ? ISQRT2 : 1.0f;
            float a2 = !v ? ISQRT2 : 1.0f;

            /* Scale according to normalizing function */
            out_data[v*8+u] = in_data[v*8+u] * a1 * a2;
        }
    }
}

static void quantize_block(float *in_data, float *out_data, uint8_t *quant_tbl)
{
    int zigzag;
    for (zigzag=0; zigzag < 64; ++zigzag)
    {
        uint8_t u = zigzag_U[zigzag];
        uint8_t v = zigzag_V[zigzag];

        float dct = in_data[v*8+u];

        /* Zig-zag and quantize */
        out_data[zigzag] = round((dct / 4.0) / quant_tbl[zigzag]);
    }
}

static void dequantize_block(float *in_data, float *out_data, uint8_t *quant_tbl)
{
    int zigzag;
    for (zigzag=0; zigzag < 64; ++zigzag)
    {
        uint8_t u = zigzag_U[zigzag];
        uint8_t v = zigzag_V[zigzag];

        float dct = in_data[zigzag];

        /* Zig-zag and de-quantize */
        out_data[v*8+u] = round((dct * quant_tbl[zigzag]) / 4.0);
    }
}

void dct_quant_block_8x8(int16_t *in_data, int16_t *out_data, uint8_t *quant_tbl)
{
    float mb[8*8] __attribute((aligned(16)));
    float mb2[8*8] __attribute((aligned(16)));

    int i, v;

    for (i=0; i<64; ++i)
        mb2[i] = in_data[i];

    for (v=0; v<8; ++v)
    {
        dct_1d(mb2+v*8, mb+v*8);
    }

    transpose_block(mb, mb2);

    for (v=0; v<8; ++v)
    {
        dct_1d(mb2+v*8, mb+v*8);
    }

    transpose_block(mb, mb2);
    scale_block(mb2, mb);
    quantize_block(mb, mb2, quant_tbl);

    for (i=0; i<64; ++i)
        out_data[i] = mb2[i];
}


void dequant_idct_block_8x8(int16_t *in_data, int16_t *out_data, uint8_t *quant_tbl)
{
    float mb[8*8] __attribute((aligned(16)));
    float mb2[8*8] __attribute((aligned(16)));

    int i, v;

    for (i=0; i<64; ++i)
        mb[i] = in_data[i];

    dequantize_block(mb, mb2, quant_tbl);

    scale_block(mb2, mb);

    for (v=0; v<8; ++v)
    {
        idct_1d(mb+v*8, mb2+v*8);
    }

    transpose_block(mb2, mb);

    for (v=0; v<8; ++v)
    {
        idct_1d(mb+v*8, mb2+v*8);
    }

    transpose_block(mb2, mb);

    for (i=0; i<64; ++i)
        out_data[i] = mb[i];
}

void catchCudaError(const char *message)
{
   hipError_t error = hipGetLastError();
   if(error!=hipSuccess) {
      fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
      exit(-1);
   }                         
}

} /* end extern "C" */


__host__
void happy_block_8x8(uint8_t *block1, uint8_t *block2, int stride, int *result)
{	
	*result = 0;
	
	const uint8_t BLOCKSIZE = 64;
		
	int *result_d = 0;
		
	uint8_t *block1_d, *block2_d;
	
	uint32_t *block1_2 = (uint32_t*) malloc(64*sizeof(uint32_t));
	uint32_t *result_block_d, *result_block_2_d;
		
	hipMalloc((void **) &block1_d, 8*stride);
	hipMalloc((void **) &block2_d, 8*stride);
	hipMalloc((void **) &result_block_d, BLOCKSIZE*sizeof(uint32_t));
	hipMalloc((void **) &result_block_2_d, BLOCKSIZE*sizeof(uint32_t));
	hipMalloc((void **) &result_d, sizeof(int));

	hipMemcpy(block1_d, block1, 8*stride, hipMemcpyHostToDevice);
	hipMemcpy(block2_d, block2, 8*stride, hipMemcpyHostToDevice);
	

	for(int i = 0; i < 8*352; i+=352) {
		for(int j = 0; j < 8; j++) {
			if(j % 8 == 0)
				printf("\n");
			printf("%3d ", block1[i+j]);
		}
	}
	printf("\n-------------------------------");
	for(int i = 0; i < 8*352; i+=352) {
		for(int j = 0; j < 8; j++) {
			if(j % 8 == 0)
				printf("\n");
			printf("%3d ", block2[i+j]);
		}
	}
	printf("\n>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>");
	happy_block_8x8_d<<<8,8>>>(block1_d, block2_d, result_block_d, stride);
	
	hipDeviceSynchronize();

	catchCudaError("Failed Execution 1");
	
	hipMemcpy(block1_2, result_block_d, 64*sizeof(uint32_t), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	
	catchCudaError("Failed Execution 2");
			
	for(int i = 0; i < 64; i++) {
			if(i % 8 == 0)
				printf("\n");
			printf("%3d ", block1_2[i]);
	}


	reduce0<<<1, 64, 64*sizeof(uint32_t)>>>(result_block_d, result_block_2_d, 64);

	hipDeviceSynchronize();

	//printf("\nresult: %d\n", *result_d);
	//printf("\n\n\n\n");

	catchCudaError("Failed Execution 3");

	hipMemcpy(result, &result_block_2_d[63], sizeof(uint32_t), hipMemcpyDeviceToHost);

	printf("\nresult: %d\n", *result);

	catchCudaError("Failed Execution 4");

	hipFree(result_block_d);
	hipFree(block1_d);
	hipFree(block2_d);
	hipFree(result_d);

}

///////////////////////////////////////////////////////////////////////////////
// CUDA KERNELS ///////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////

__global__
void happy_block_8x8_d(uint8_t *block1_d, uint8_t *block2_d, uint32_t *result_block_d, int stride)
{
	int i = blockIdx.x * stride + threadIdx.x;
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	if(j < 64) {
		result_block_d[j] = abs(block1_d[i] - block2_d[i]);

	}
}

__global__
void diff_abs_frame(uint8_t *block1_d, uint8_t *block2_d, uint8_t *result_block_d, int size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < size) {
		result_block_d[i] = abs(block1_d[i] - block2_d[i]);

	}
}


///////////////////////////////////////////////////////////////////////////////

__global__
void reduce0(uint32_t *g_idata, uint32_t *g_odata, uint32_t n) {

    extern __shared__ uint32_t temp[];
    int thid = threadIdx.x;

    temp[thid] = g_idata[thid];

    __syncthreads();

    for(int offset = 1;offset < n; offset *= 2) {
        if(thid >= offset)
            temp[thid] += temp[thid - offset];

        __syncthreads();
    }

   	g_odata[thid] = temp[thid];
}

