#include "hip/hip_runtime.h"
extern "C" {

#include <inttypes.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include "tables.h"

#define ISQRT2 0.70710678118654f

static void transpose_block(float *in_data, float *out_data)
{
    int i,j;
    for (i=0; i<8; ++i)
        for (j=0; j<8; ++j)
        {
            out_data[i*8+j] = in_data[j*8+i];
        }
}

static void dct_1d(float *in_data, float *out_data)
{
    int i,j;

    for (j=0; j<8; ++j)
    {
        float dct = 0;

        for (i=0; i<8; ++i)
        {
            dct += in_data[i] * dctlookup[i][j];
        }

        out_data[j] = dct;
    }
}

static void idct_1d(float *in_data, float *out_data)
{
    int i,j;

    for (j=0; j<8; ++j)
    {
        float idct = 0;

        for (i=0; i<8; ++i)
        {
            idct += in_data[i] * dctlookup[j][i];
        }

        out_data[j] = idct;
    }
}


static void scale_block(float *in_data, float *out_data)
{
    int u,v;

    for (v=0; v<8; ++v)
    {
        for (u=0; u<8; ++u)
        {
            float a1 = !u ? ISQRT2 : 1.0f;
            float a2 = !v ? ISQRT2 : 1.0f;

            /* Scale according to normalizing function */
            out_data[v*8+u] = in_data[v*8+u] * a1 * a2;
        }
    }
}

static void quantize_block(float *in_data, float *out_data, uint8_t *quant_tbl)
{
    int zigzag;
    for (zigzag=0; zigzag < 64; ++zigzag)
    {
        uint8_t u = zigzag_U[zigzag];
        uint8_t v = zigzag_V[zigzag];

        float dct = in_data[v*8+u];

        /* Zig-zag and quantize */
        out_data[zigzag] = round((dct / 4.0) / quant_tbl[zigzag]);
    }
}

static void dequantize_block(float *in_data, float *out_data, uint8_t *quant_tbl)
{
    int zigzag;
    for (zigzag=0; zigzag < 64; ++zigzag)
    {
        uint8_t u = zigzag_U[zigzag];
        uint8_t v = zigzag_V[zigzag];

        float dct = in_data[zigzag];

        /* Zig-zag and de-quantize */
        out_data[v*8+u] = round((dct * quant_tbl[zigzag]) / 4.0);
    }
}

void dct_quant_block_8x8(int16_t *in_data, int16_t *out_data, uint8_t *quant_tbl)
{
    float mb[8*8] __attribute((aligned(16)));
    float mb2[8*8] __attribute((aligned(16)));

    int i, v;

    for (i=0; i<64; ++i)
        mb2[i] = in_data[i];

    for (v=0; v<8; ++v)
    {
        dct_1d(mb2+v*8, mb+v*8);
    }

    transpose_block(mb, mb2);

    for (v=0; v<8; ++v)
    {
        dct_1d(mb2+v*8, mb+v*8);
    }

    transpose_block(mb, mb2);
    scale_block(mb2, mb);
    quantize_block(mb, mb2, quant_tbl);

    for (i=0; i<64; ++i)
        out_data[i] = mb2[i];
}


void dequant_idct_block_8x8(int16_t *in_data, int16_t *out_data, uint8_t *quant_tbl)
{
    float mb[8*8] __attribute((aligned(16)));
    float mb2[8*8] __attribute((aligned(16)));

    int i, v;

    for (i=0; i<64; ++i)
        mb[i] = in_data[i];

    dequantize_block(mb, mb2, quant_tbl);

    scale_block(mb2, mb);

    for (v=0; v<8; ++v)
    {
        idct_1d(mb+v*8, mb2+v*8);
    }

    transpose_block(mb2, mb);

    for (v=0; v<8; ++v)
    {
        idct_1d(mb+v*8, mb2+v*8);
    }

    transpose_block(mb2, mb);

    for (i=0; i<64; ++i)
        out_data[i] = mb[i];
}

extern __host__ void happy_block_8x8(uint8_t *block1, uint8_t *block2, int stride, int *result);
extern __global__ void happy_block_8x8_d(uint8_t *block1, uint8_t *block2, uint32_t *result_block_d, int stride);
extern __global__ void reduce_sum(uint32_t *result_block_d, int *result_d);


void cuda_error(const char *message)
{
   hipError_t error = hipGetLastError();
   if(error!=hipSuccess) {
      fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
      exit(-1);
   }                         
}

} /* end extern "C" */


__host__
void happy_block_8x8(uint8_t *block1, uint8_t *block2, int stride, int *result)
{	
	*result = 0;
	
	const uint8_t BLOCKSIZE = 64;
		
	int *result_d = 0;
		
	uint8_t *block1_d, *block2_d;
	
	uint32_t *result_block_d;
		
	hipMalloc((void **) &block1_d, 8*stride);
	hipMalloc((void **) &block2_d, 8*stride);
	hipMalloc((void **) &result_block_d, BLOCKSIZE*sizeof(uint32_t));
	hipMalloc((void **) &result_d, sizeof(int));
/*	
	hipMemcpy2D((void *) block1_d, 8, (const void *) block1, stride, 8, 8, hipMemcpyHostToDevice);
	hipMemcpy2D((void *) block2_d, 8, (const void *) block2, stride, 8, 8, hipMemcpyHostToDevice);
*/
	hipMemcpy(block1_d, block1, 8*stride, hipMemcpyHostToDevice);
	hipMemcpy(block2_d, block2, 8*stride, hipMemcpyHostToDevice);

	for(int i = 0; i < 8*352; i+=352) {
		for(int j = 0; j < 8; j++) {
			if(j % 8 == 0)
				printf("\n");
			printf("%3d ", block1[i+j]);
		}
	}
	
	printf("\n----");
	
	for(int i = 0; i < 8*352; i+=352) {
		for(int j = 0; j < 8; j++) {
			if(j % 8 == 0)
				printf("\n");
			printf("%3d ", block1_d[i+j]);
		}
	}
	
	printf("\n\n\n\n");

			
	happy_block_8x8_d<<<8, 8>>>(block1_d, block2_d, result_block_d, stride);
		
	hipDeviceSynchronize();
	
	reduce_sum<<<1, 64>>>(result_block_d, result_d);
	
	hipDeviceSynchronize();
	
	hipMemcpy((void *) result, result_d, sizeof(int), hipMemcpyDeviceToHost);	
	
	hipFree(result_block_d);
	hipFree(block1_d);
	hipFree(block2_d);	
	hipFree(result_d);

}

__global__
void reduce_sum(uint32_t *result_block_d, int *result_d)
{
	__shared__ uint32_t sdata[64];
	
	uint32_t tid = threadIdx.x;
    uint32_t i = blockIdx.x*blockDim.x + threadIdx.x;

	sdata[i] = result_block_d[i] + result_block_d[i+blockDim.x];
	
	__syncthreads();
	
	for(uint32_t s = blockDim.x/2; s>0; s>>=1) 
	{
    	if (tid < s) 
	    {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
   	}
	
	if(tid == 0)
		*result_d = sdata[tid];
}

__global__ 
void happy_block_8x8_d(uint8_t *block1_d, uint8_t *block2_d, uint32_t *result_block_d, int stride)
{
	int i = blockIdx.x * stride + threadIdx.x;
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(i < 64) {
		result_block_d[j] = abs(block1_d[i] - block2_d[i]);	
	}	
}
