#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <getopt.h>
#include <errno.h>
#include <stdint.h>
#include <math.h>
#include <assert.h>
#include <limits.h>
#include "c63.h"
#include "me.hcu"
#include "dsp.hcu"

extern "C" {

/* Motion estimation for 8x8 block */

void testEqual(uint16_t *results, uint8_t *ref,int width,int height, int result_stride, int ref_stride) {
	uint16_t *res_host = (uint16_t*)malloc(height*result_stride*sizeof(uint16_t));
	uint8_t *ref_host = (uint8_t*)malloc(height*ref_stride*sizeof(uint8_t));

	hipMemcpy(res_host,results, height*result_stride*sizeof(uint16_t), hipMemcpyDeviceToHost);
	hipMemcpy(ref_host,ref, height*ref_stride*sizeof(uint8_t), hipMemcpyDeviceToHost);
	int x = 0, y = 0;
	for(y = 0; y < height; y++) {
		printf("\nROW %d= ", y);
		for(x = 0; x < width;x++) {
			const char *c = ref_host[y*ref_stride+x] == res_host[y*result_stride+x] ? "C" : "W";
			printf("%s",c);
		}
	}
	printf("\n");
	free(res_host);
	free(ref_host);
	exit(-1);
}


void printResult(uint16_t *results,int width,int height, int result_stride) {
	uint16_t *res_host = (uint16_t*)malloc(height*result_stride*sizeof(uint16_t));

	hipMemcpy(res_host,results, height*result_stride*sizeof(uint16_t), hipMemcpyDeviceToHost);
	int x = 0, y = 0;
	for(y = 0; y < height; y++) {
		printf("ROW %d= ", y);
		for(x = 0; x < width;x++) {
			printf("%d, ", res_host[y*result_stride+x]);
		}
		printf("\n");
	}
	free(res_host);
}


void me_block_8x8(struct c63_common *cm, int mb_x, int mb_y, uint8_t *orig, uint8_t *ref, int cc,uint16_t *result_sad, uint16_t *sums)
{
    struct macroblock *mb = &cm->curframe->mbs[cc][mb_y * cm->padw[cc]/8 + mb_x];
    int range = cm->me_search_range;

    int left = mb_x*8 - range;
    int top = mb_y*8 - range;
    int right = mb_x*8 + range;
    int bottom = mb_y*8 + range;

    int w = cm->padw[cc];
    int h = cm->padh[cc];

    /* Make sure we are within bounds of reference frame */
    // TODO: Support partial frame bounds
    bool rightEdge = false;
    bool bottomEdge = false;
    if (left < 0)
        left = 0;
    if (top < 0)
        top = 0;
    if (right > (w - 8)) {
        right = w - 8;
        rightEdge = true;
    }
    if (bottom > (h - 8)) {
        bottom = h - 8;
        bottomEdge = true;
    }


    int x,y;
    int mx = mb_x * 8;
    int my = mb_y * 8;

    int best_sad = INT_MAX;
    int search_width = right-left;
    int search_height = bottom-top;
    int num_mb_width = search_width / 8;
    int num_mb_height = search_height / 8;


    dim3 threadSize(8,8,4);

    cuda_me<<<1,threadSize>>>(orig + my*w+mx, ref + top*w+left, w, result_sad, num_mb_width,num_mb_height,rightEdge, bottomEdge);

    catchCudaError("FAILED happy_block_8x8");

	//print_buffer32(result_block1, 64*64);
	hipMemcpy(sums,result_sad, 40*40 * sizeof(uint16_t), hipMemcpyDeviceToHost);
	//#pragma unroll loop 64

    for (y=0; y<search_height; ++y)
    {
        for (x=0; x<search_width; ++x)
        {
            int sad = sums[y*40+x];
            //sad_block_8x8(orig + my*w+mx, ref + y*w+x, w, &sad);

            //printf("(%4d,%4d) %d\n", x, y, sad);

            if (sad < best_sad)
            {
                mb->mv_x = left + x - mx;
                mb->mv_y = top + y - my;
                best_sad = sad;
                //printf("new best sad for (%d,%d) @ (%4d,%4d) = %d\n", mb_x, mb_y, x, y, sad);
            }
        }
    }

    /* Here, there should be a threshold on SAD that checks if the motion vector is
     * cheaper than intraprediction. We always assume MV to be beneficial */

//    printf("Using motion vector (%d, %d) with SAD %d\n", mb->mv_x, mb->mv_y, best_sad);

    mb->use_mv = 1;
}

extern "C" void c63_motion_estimate(struct c63_common *cm)
{
    /* Compare this frame with previous reconstructed frame */
	uint8_t *image_orig, *image_ref, *image_orig_2, *image_ref_2;
	int size = cm->width*cm->height;

    uint16_t *result_sad, *sums;
    hipMalloc((void**)&result_sad, 40*40 * sizeof(uint16_t));
	hipMalloc((void**)&image_orig, size * sizeof(uint8_t));
	hipMalloc((void**)&image_ref, size * sizeof(uint8_t));
	sums = (uint16_t*)malloc(40*40 * sizeof(uint16_t));

	hipMemcpy(image_orig, cm->curframe->orig->Y, size * sizeof(uint8_t), hipMemcpyHostToDevice);
	hipMemcpy(image_ref, cm->refframe->recons->Y, size * sizeof(uint8_t), hipMemcpyHostToDevice);

    // Luma
	int mb_y, mb_x;
    for (mb_y=0; mb_y < cm->mb_rows; ++mb_y)
    {
        for (mb_x=0; mb_x < cm->mb_cols; ++mb_x)
        {
            me_block_8x8(cm, mb_x, mb_y, image_orig, image_ref, 0, result_sad, sums);
        }
    }

    hipMalloc((void**)&image_orig_2, size * sizeof(uint8_t));
   	hipMalloc((void**)&image_ref_2, size * sizeof(uint8_t));

	hipMemcpy(image_orig, cm->curframe->orig->U, size * sizeof(uint8_t), hipMemcpyHostToDevice);
	hipMemcpy(image_ref, cm->refframe->recons->U, size * sizeof(uint8_t), hipMemcpyHostToDevice);
	hipMemcpy(image_orig_2, cm->curframe->orig->V, size * sizeof(uint8_t), hipMemcpyHostToDevice);
	hipMemcpy(image_ref_2, cm->refframe->recons->V, size * sizeof(uint8_t), hipMemcpyHostToDevice);

    // Chroma
    for (mb_y=0; mb_y < cm->mb_rows/2; ++mb_y)
    {
        for (mb_x=0; mb_x < cm->mb_cols/2; ++mb_x)
        {
            me_block_8x8(cm, mb_x, mb_y, image_orig, image_ref, 1, result_sad,sums);
            me_block_8x8(cm, mb_x, mb_y, image_orig_2, image_ref_2, 2,result_sad,sums);
        }
    }

    hipFree(image_orig);
    hipFree(image_orig_2);
    hipFree(image_ref);
    hipFree(image_ref_2);
    hipFree(result_sad);
    free(sums);

}

/* Motion compensation for 8x8 block */
__host__
void mc_block_8x8(struct c63_common *cm, int mb_x, int mb_y, uint8_t *predicted, uint8_t *ref, int cc)
{
    struct macroblock *mb = &cm->curframe->mbs[cc][mb_y * cm->padw[cc]/8 + mb_x];

    if (!mb->use_mv)
        return;

    int left = mb_x*8;
    int top = mb_y*8;
    int right = left + 8;
    int bottom = top + 8;

    int w = cm->padw[cc];

    /* Copy block from ref mandated by MV */
    int x,y;
    for (y=top; y < bottom; ++y)
    {
        for (x=left; x < right; ++x)
        {
            predicted[y*w+x] = ref[(y + mb->mv_y) * w + (x + mb->mv_x)];
        }
    }
}

extern void c63_motion_compensate(struct c63_common *cm)
{
    int mb_x, mb_y;

    /* Luma */
    for (mb_y=0; mb_y < cm->mb_rows; ++mb_y)
    {
        for (mb_x=0; mb_x < cm->mb_cols; ++mb_x)
        {
            mc_block_8x8(cm, mb_x, mb_y, cm->curframe->predicted->Y, cm->refframe->recons->Y, 0);
        }
    }

    /* Chroma */
    for (mb_y=0; mb_y < cm->mb_rows/2; ++mb_y)
    {
        for (mb_x=0; mb_x < cm->mb_cols/2; ++mb_x)
        {
            mc_block_8x8(cm, mb_x, mb_y, cm->curframe->predicted->U, cm->refframe->recons->U, 1);
            mc_block_8x8(cm, mb_x, mb_y, cm->curframe->predicted->V, cm->refframe->recons->V, 2);
        }
    }
}

}
